#include "hip/hip_runtime.h"
// GPU/CPU Convolution engine
//  GPUconv.cu
//  GPUconv
//
//  Created by Davide Andrea Mauro on 2011-07-29.
//	Last Edited by Davide Andrea Mauro on 2013-02-27.
//

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <shrQATest.h>
#include <hipfft/hipfft.h>

//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_cuda_drvapi.h>
#include <helper_functions.h>

#include <GPUconv.cuh>


static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex, hipfftComplex);
static __global__ void ComplexPointwiseMul(hipfftComplex*, const hipfftComplex* , int size);

float GPUconv(float* input, int SIGNAL_SIZE, float* filtersx, float* filterdx, int FILTER_KERNEL_SIZE, float* outputsx, float* outputdx, int direct) 
{
	//Look for CUDA capable Devices
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	printf("%d.\n", deviceCount);
	if (deviceCount == 0){
		printf("There is no device supporting CUDA.\n");
		return -1.0f;
	}
	//hipDeviceReset();

	//Pick the best one
	//int rId=cutGetMaxGflopsDeviceId();
	int rId=gpuGetMaxGflopsDeviceId();
	hipSetDevice(rId);
	//Pick properties used for block and grid
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, rId);

	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);
	//printf("Free/Total %lu/%lu %u %%\n",free_mem, total_mem, (unsigned int) ((free_mem*100.0)/total_mem));	
	//hipError_t error = cudaThreadSetLimit(hipLimitMallocHeapSize, (size_t) (0.8 * free_mem));

	if(direct==1){
		//Work on it! Caution Memory limit
		int required_mem=3*((SIGNAL_SIZE+FILTER_KERNEL_SIZE-1)*sizeof(hipfftComplex));
		//printf("Required Memory: %d.\n", required_mem);
		if (required_mem>=(free_mem*0.8)){
			printf("Insufficient memory on device. Required %d, Available %lu.\n", required_mem, free_mem);
			return -1.0f;
		}

		int new_size=SIGNAL_SIZE+FILTER_KERNEL_SIZE-1;
		int mem_size=sizeof(hipfftComplex) *new_size;
		printf("Entering Direct Mode.\n");

		//First thing to do: PAD!
		hipfftComplex* h_signal = (hipfftComplex*)malloc(mem_size);
		// Initalize the memory for the signal
		for (int i = 0; i < new_size; i++) {
			if (i<SIGNAL_SIZE){
				h_signal[i].x = input[i];
			}
			else{
				h_signal[i].x = 0.0f;
			}
			h_signal[i].y = 0.0f;
		}

		hipfftComplex* h_filter_kernels[2];
		for (int k=0; k<2;k++){
			h_filter_kernels[k]=(hipfftComplex*)malloc(mem_size);
		}
		// Initalize the memory for the filter
		for (int i = 0; i < new_size; i++) {
			if(i<FILTER_KERNEL_SIZE){
				h_filter_kernels[0][i].x = filtersx[i];
				h_filter_kernels[1][i].x = filterdx[i];
			}
			else{
				h_filter_kernels[0][i].x = 0.0f;
				h_filter_kernels[1][i].x = 0.0f;
			}
			h_filter_kernels[0][i].y = 0.0f;
			h_filter_kernels[1][i].y = 0.0f;
		}


		// CUFFT plan
		hipfftHandle plan;	
		int window=new_size;
		//cufftSafeCall(hipfftPlan1d(&plan, window, HIPFFT_C2C, (new_size/window)));
		hipfftPlan1d(&plan, window, HIPFFT_C2C, (new_size/window));

		// Allocate device memory for signal
		hipfftComplex* d_signal;
		//cutilSafeCall(hipMalloc((void**)&d_signal, mem_size));
		hipMalloc((void**)&d_signal, mem_size);

		// Copy host memory to device
		//cutilSafeCall(hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice));
		hipMemcpy(d_signal, h_signal, mem_size, hipMemcpyHostToDevice);
		//   printf("Device Memory allocated for Signal.\n");


		// Allocate device memory for filter kernel
		hipfftComplex* d_filter_kernels[2];
		for (int i=0; i<2;i++){
			//cutilSafeCall(hipMalloc((void**)&d_filter_kernels[i], mem_size));
			//// Copy host memory to device
			//cutilSafeCall(hipMemcpy(d_filter_kernels[i], h_filter_kernels[i], mem_size, hipMemcpyHostToDevice));
			hipMalloc((void**)&d_filter_kernels[i], mem_size);
			// Copy host memory to device
			hipMemcpy(d_filter_kernels[i], h_filter_kernels[i], mem_size, hipMemcpyHostToDevice);
		}
		// printf("Device Memory allocated for Filters.\n");


		//	printf("Transforming signal hipfftExecC2C\n");
		//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD));
		hipfftExecC2C(plan, (hipfftComplex *)d_signal, (hipfftComplex *)d_signal, HIPFFT_FORWARD);

		for (int i=0; i<2;i++){
			//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_FORWARD));
			hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_FORWARD);

		}
		hipDeviceSynchronize();


		// Multiply the coefficients together and normalize the result
		int block_size = 256;//deviceProp.maxThreadsPerBlock; // 
		int grid_size = new_size/256 ;//deviceProp.warpSize; //

		for (int i=0; i<2;i++)
			ComplexPointwiseMul<<<grid_size, block_size>>>(d_filter_kernels[i], d_signal, new_size);
		hipDeviceSynchronize();
		// Check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed [ ComplexPointwiseMul ]");


		// Transform signal back
		for (int i=0; i<2;i++){
			//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_BACKWARD));
			hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_BACKWARD);

		}
		hipDeviceSynchronize();

		// Copy device memory to host
		hipfftComplex* h_convolved_signal[2];
		for (int i=0; i<2;i++){
			h_convolved_signal[i]= (hipfftComplex*)malloc(mem_size);
			//cutilSafeCall(hipMemcpy(h_convolved_signal[i], d_filter_kernels[i], mem_size, hipMemcpyDeviceToHost));
			hipMemcpy(h_convolved_signal[i], d_filter_kernels[i], mem_size, hipMemcpyDeviceToHost);
		}


		//printf("Writing back.\n");
		//outputsx=(float*)malloc(sizeof(float) * new_size);
		//outputdx=(float*)malloc(sizeof(float) * new_size);

		float maxo[2];
		maxo[0]=0.0f;
		maxo[1]=0.0f;  
		for (int i = 0; i < new_size; i++){
			if (abs(maxo[0])<=abs(h_convolved_signal[0][i].x)) maxo[0]=h_convolved_signal[0][i].x;
			if (abs(maxo[1])<=abs(h_convolved_signal[1][i].x)) maxo[1]=h_convolved_signal[1][i].x;
		}
		float maxot= abs(maxo[0])>=abs(maxo[1])? abs(maxo[0]): abs(maxo[1]);
		for (int i = 0; i < new_size; i++) {
			outputsx[i]=(h_convolved_signal[0][i].x/maxot);
			outputdx[i]=(h_convolved_signal[1][i].x/maxot);
		}

		//printf("Freeing resources.\n");
		//Destroy CUFFT context
		//cufftSafeCall(hipfftDestroy(plan));
		hipfftDestroy(plan);
		
		// cleanup memory
		free(h_signal);
		//cutilSafeCall(hipFree(d_signal));
		hipFree(d_signal);
		
		for (int i=0; i<2;i++){
			free(h_filter_kernels[i]);
			free(h_convolved_signal[i]);
			//cutilSafeCall(hipFree(d_filter_kernels[i]));
			hipFree(d_filter_kernels[i]);
		}
		hipDeviceSynchronize();
		//cutilDeviceReset(); 
		hipDeviceReset(); 
		return new_size;
	}


	if(direct==0){
		//Work on it! Caution Memory limit
		int required_mem=3*((FILTER_KERNEL_SIZE*2)*sizeof(hipfftComplex));
		//printf("Required Memory: %d.\n", required_mem);
		if (required_mem>=(free_mem*0.8)){
			printf("Insufficient memory on device. Required %d, Available %lu.\n", required_mem, free_mem);
			return -1.0f;
		}
		printf("Entering Overlap and Save Mode.\n");

		int new_size=SIGNAL_SIZE+FILTER_KERNEL_SIZE-1;
		int mem_size=sizeof(hipfftComplex) *new_size;
		int payload=FILTER_KERNEL_SIZE*2;
		int mem_pay= sizeof(hipfftComplex)*payload;


		hipfftComplex* h_filter_kernels[2];
		for (int k=0; k<2;k++){
			h_filter_kernels[k]=(hipfftComplex*)malloc(mem_pay);
		}
		// Initalize the memory for the filter
		for (int i = 0; i < payload; i++) {
			if(i<FILTER_KERNEL_SIZE){
				h_filter_kernels[0][i].x = filtersx[i];
				h_filter_kernels[1][i].x = filterdx[i];
			}
			else{
				h_filter_kernels[0][i].x = 0.0f;
				h_filter_kernels[1][i].x = 0.0f;
			}
			h_filter_kernels[0][i].y = 0.0f;
			h_filter_kernels[1][i].y = 0.0f;
		}

		// CUFFT plan
		hipfftHandle plan;	
		int window=payload;
		//cufftSafeCall(hipfftPlan1d(&plan, window, HIPFFT_C2C, (payload/window)));
		hipfftPlan1d(&plan, window, HIPFFT_C2C, (payload/window));
		// Allocate device memory for filter kernel
		hipfftComplex* d_filter_kernels[2];
		for (int i=0; i<2;i++){
			//cutilSafeCall(hipMalloc((void**)&d_filter_kernels[i], mem_pay));
			hipMalloc((void**)&d_filter_kernels[i], mem_pay);
			// Copy host memory to device
			//cutilSafeCall(hipMemcpy(d_filter_kernels[i], h_filter_kernels[i], mem_pay, hipMemcpyHostToDevice));
			hipMemcpy(d_filter_kernels[i], h_filter_kernels[i], mem_pay, hipMemcpyHostToDevice);
		}
		// printf("Device Memory allocated for Filters.\n");
		//	printf("Transforming signal hipfftExecC2C\n");
		hipDeviceSynchronize();
		for (int i=0; i<2;i++){
			//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_FORWARD));
			hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernels[i], (hipfftComplex *)d_filter_kernels[i], HIPFFT_FORWARD);
		}
		hipDeviceSynchronize();


		float maxo[2];
		maxo[0]=0.0f;
		maxo[1]=0.0f; 

		float* tresultsx=(float*)malloc(sizeof(float) * (SIGNAL_SIZE+(FILTER_KERNEL_SIZE*2)));
		float* tresultdx=(float*)malloc(sizeof(float) * (SIGNAL_SIZE+(FILTER_KERNEL_SIZE*2)));

		for(int j=0; j<(SIGNAL_SIZE+(FILTER_KERNEL_SIZE*2)); j++){ 
			tresultsx[j]=0.0f;
			tresultdx[j]=0.0f;
		}

		//BIG Loop
		for(int k=0; k<SIGNAL_SIZE; (k+=FILTER_KERNEL_SIZE)){
			//printf("%d.\n",k);
			//First thing to do: PAD!
			hipfftComplex* h_signal = (hipfftComplex*)malloc(mem_pay);
			// Initalize the memory for the signal
			for (int i = 0; i < payload; i++) {
				if (((k+i)<SIGNAL_SIZE)&&(i<(payload/2))){
					h_signal[i].x = input[k+i];
				}
				else{
					h_signal[i].x = 0.0f;
				}
				h_signal[i].y = 0.0f;
			}
			// Allocate device memory for signal
			hipfftComplex* d_signal[2];
			for (int i=0; i<2;i++){
				//cutilSafeCall(hipMalloc((void**)&d_signal[i], mem_pay));
				hipMalloc((void**)&d_signal[i], mem_pay);
				// Copy host memory to device
				//cutilSafeCall(hipMemcpy(d_signal[i], h_signal, mem_pay, hipMemcpyHostToDevice));
				hipMemcpy(d_signal[i], h_signal, mem_pay, hipMemcpyHostToDevice);

				//   printf("Device Memory allocated for Signal.\n");
				//	printf("Transforming signal hipfftExecC2C\n");
				//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_signal[i], (hipfftComplex *)d_signal[i], HIPFFT_FORWARD));
				hipfftExecC2C(plan, (hipfftComplex *)d_signal[i], (hipfftComplex *)d_signal[i], HIPFFT_FORWARD);
			}
			hipDeviceSynchronize();
			// Multiply the coefficients together and normalize the result
			int block_size = deviceProp.maxThreadsPerBlock; // 256;//
			int grid_size = deviceProp.warpSize; //new_size/256 ;//

			for (int i=0; i<2;i++){
				ComplexPointwiseMul<<<grid_size, block_size>>>(d_signal[i], d_filter_kernels[i], payload);
				// Check if kernel execution generated and error
				//cutilCheckMsg("Kernel execution failed [ ComplexPointwiseMul ]");
			}
			hipDeviceSynchronize();
			// Transform signal back
			for (int i=0; i<2;i++){
				//cufftSafeCall(hipfftExecC2C(plan, (hipfftComplex *)d_signal[i], (hipfftComplex *)d_signal[i], HIPFFT_BACKWARD));
				hipfftExecC2C(plan, (hipfftComplex *)d_signal[i], (hipfftComplex *)d_signal[i], HIPFFT_BACKWARD);
			}
			hipDeviceSynchronize();


			// Copy device memory to host
			hipfftComplex* h_convolved_signal[2];
			for (int i=0; i<2;i++){
				h_convolved_signal[i]= (hipfftComplex*)malloc(mem_pay);
				//cutilSafeCall(hipMemcpy(h_convolved_signal[i], d_signal[i], mem_pay, hipMemcpyDeviceToHost));
				hipMemcpy(h_convolved_signal[i], d_signal[i], mem_pay, hipMemcpyDeviceToHost);
			}
			//printf("Writing back.\n");
			for (int i=0; i< payload; i++){
				tresultsx[k+i]+=h_convolved_signal[0][i].x;
				tresultdx[k+i]+=h_convolved_signal[1][i].x;
				if (abs(maxo[0])<=abs(tresultsx[k+i])) maxo[0]=tresultsx[k+i];
				if (abs(maxo[1])<=abs(tresultdx[k+i])) maxo[1]=tresultdx[k+i];
			}

			free(h_signal);
			free(h_convolved_signal[0]);
			free(h_convolved_signal[1]);
			//cutilSafeCall(hipFree(d_signal[0]));
			//cutilSafeCall(hipFree(d_signal[1]));
			hipFree(d_signal[0]);
			hipFree(d_signal[1]);
		}

		float maxot= abs(maxo[0])>=abs(maxo[1])? abs(maxo[0]): abs(maxo[1]);
		for (int i = 0; i < new_size; i++) {
			outputsx[i]=(tresultsx[i]/maxot);
			outputdx[i]=(tresultdx[i]/maxot);
		}

		//printf("Freeing resources.\n");
		//Destroy CUFFT context
		//cufftSafeCall(hipfftDestroy(plan));
		hipfftDestroy(plan);
		// cleanup memory
		free(h_filter_kernels[0]);
		free(h_filter_kernels[1]);
		//cutilSafeCall(hipFree(d_filter_kernels[0]));
		//cutilSafeCall(hipFree(d_filter_kernels[1]));
		hipFree(d_filter_kernels[0]);
		hipFree(d_filter_kernels[1]);
		free(tresultsx);
		free(tresultdx);
		hipDeviceSynchronize();
		//cutilDeviceReset(); 
		hipDeviceReset(); 


		return new_size;
	}

	return -1.0f;
}


////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex c;  
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

// Complex pointwise multiplication
// Based on ComplexPointwiseMulAndScale but without scaling... It creates more problems than it solves...
static __global__ void ComplexPointwiseMul(hipfftComplex* a, const hipfftComplex* b, int size)
{
	const int numThreads = blockDim.x * gridDim.x;
	const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = threadID; i < size; i += numThreads)
		a[i] =ComplexMul(a[i], b[i]); 
} 